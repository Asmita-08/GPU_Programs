#include<hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  //Device ID is required first to query the device.

  int deviceId;
  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  //The following contains various properties about the current device.

  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;


  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}