#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void shared_memory_example(int *a, int *b, int n) {
    __shared__ int sharedData[256];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        sharedData[threadIdx.x] = a[idx];
        __syncthreads();
        b[idx] = sharedData[threadIdx.x] * 2;
    }
}

int main() {
    int n = 256;
    int *h_a, *h_b, *d_a, *d_b;

    size_t size = n * sizeof(int);
    
    // Allocate memory
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);

    // Initialize input
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
    }

    // Copy to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

    // Launch kernel
    shared_memory_example<<<1, n>>>(d_a, d_b, n);

    // Copy back result
    hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);

    // Print results
    printf("Shared Memory Example Output:\n");
    for (int i = 0; i < 10; i++) {
        printf("%d ", h_b[i]);
    }
    printf("\n");

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

    return 0;
}
