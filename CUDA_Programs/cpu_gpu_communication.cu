#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void myKernel(char *arr, int l){
        unsigned id = threadIdx.x;
        if(id<l){
                ++arr[id];
        }
}

int main(){
        char cpuarr[]="abcdefgh12345678";
        char *gpuarr;

        hipMalloc(&gpuarr, sizeof(char) *(1 + strlen(cpuarr)));

        hipMemcpy(gpuarr, cpuarr, sizeof(char) *(1 + strlen(cpuarr)), hipMemcpyHostToDevice);

        myKernel<<<1, 32>>>(gpuarr, strlen(cpuarr) + 1);

        hipDeviceSynchronize();

	hipMemcpy(cpuarr, gpuarr, sizeof(char) *(1 + strlen(cpuarr)), hipMemcpyDeviceToHost);

        printf(cpuarr);
        return 0;
}


