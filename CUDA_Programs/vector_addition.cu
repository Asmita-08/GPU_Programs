#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 256;
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    size_t size = n * sizeof(int);

    // Allocate host memory
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Define grid and block sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print some results
    printf("Vector Addition Result:\n");
    for (int i = 0; i < 10; i++) {
        printf("%d ", h_c[i]);
    }
    printf("\n");

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
