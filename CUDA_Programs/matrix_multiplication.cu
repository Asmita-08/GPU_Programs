#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 3  

__global__ void matrix_mult(int *A, int *B, int *C, int n) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int h_A[N * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int h_B[N * N] = {9, 8, 7, 6, 5, 4, 3, 2, 1};
    int h_C[N * N];

    int *d_A, *d_B, *d_C;
    int size = N * N * sizeof(int);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    matrix_mult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Matrix Multiplication Result:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", h_C[i * N + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
