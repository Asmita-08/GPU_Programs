#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 100

__global__ void squareOfArrayElements(int *a){
        a[threadIdx.x]=threadIdx.x*threadIdx.x;
}

int main(){
        int a[N], *da;

        hipMalloc(&da, N *sizeof(int));
        squareOfArrayElements<<<1, N>>>(da);

        hipMemcpy(a, da, N *sizeof(int), hipMemcpyDeviceToHost);

        for(int i=0; i<N; ++i){
                printf("%d \n", a[i]);
        }
        return 0;
}

